#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""


#include<iostream>
#include <stdio.h>
#include <chrono>
#include<time.h>
using namespace std;
#define N 128 //количество элементов в массиве

void add_cpu(int* a, int* b, int* c)
{
    int tid = 0;
    while (tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid += 1;
    }
}

__global__ void add_gpu(int* a, int* b, int* c)
{
    int tid = threadIdx.x; // индекс элемента
    if (tid > N - 1)// проверка за пределы массива
        return;// 
    c[tid] = a[tid] + b[tid];// сложение массивов
}
int main()
{
    int a[N];// память на CPU
    int b[N];
    int c[N];


    for (int i = 0; i < N; i++) //заполнение массива
    {
        a[i] = i*i;
        b[i] = -i;
    }

    //long before = time(NULL);
    auto before = std::chrono::steady_clock::now();
    add_cpu(a, b, c);
    auto after = std::chrono::steady_clock::now();
    auto elapsed_ms = std::chrono::duration_cast<std::chrono::milliseconds>(after - before);
    //long after = time(NULL);


    for (int i = 0; i < N; i++)
    {
        printf("%d+%d=%d\n", a[i], b[i], c[i]);
    }
    // память на GPU
    int* dev_a;
    int* dev_b;
    int* dev_c;

    hipMalloc((void**)&dev_a, N * sizeof(int));// память на GPU
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice); //копирование данных GPU
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    float gpuTime = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    add_gpu << <1, N >> > (dev_a, dev_b, dev_c); //вызов ядра

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("time on gpu=%f ms\n", gpuTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice);//результат с GPU
    for (int i = 0; i < N; i++)// вывод рассчетов
    {
        printf("%d+%d=%d\n", a[i], b[i], c[i]);
    }
    printf("time on gpu=%.10f ms\n", gpuTime);// время GPU
    cout << "time on cpu=" << elapsed_ms.count() << "ms";
    //printf("time on cpu=&lf ms\n", elapsed_ms); //время CPU неудалось
    hipFree(dev_a);//освобождение памяти GPU
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}